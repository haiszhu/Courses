#include "hip/hip_runtime.h"
/*
 * cuda lap slp pot & matlab mex interface
 * 11/15/23 Hai, do not form matrix explicitly
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
// #include "hipblas.h"
/* these are for matlab */
#include "mex.h"
#include "gpu/mxGPUArray.h"

/*
 * Device code
 */
void __global__ culapslppot(double const * const src,
                            double const * const targ,
                            double const * const x,
                            double * const y,
                            int const N,
                            int const M)
{
  /* Calculate the global linear index, assuming a 1-d grid. */
  int const i = blockDim.x * blockIdx.x + threadIdx.x;
  double dx, dy, dz;
  if (i<M) {
    y[i] = 0.0;
    for (int j = 0; j < N; ++j) {
      dx = src[3*j]   - targ[3*i];
      dy = src[3*j+1] - targ[3*i+1];
      dz = src[3*j+2] - targ[3*i+2];
      y[i] += x[j]/sqrt(dx*dx+dy*dy+dz*dz);
    }
  }
}

/*
 * Host code (just slp potential)
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxDouble *src, *targ, *x;  /* input: source, target, density */
    mxDouble *y;               /* output: potential */
    double *d_src, *d_targ;
    double *d_x, *d_y;
    //double *d_A;
    int N, M;                  /* dimension */
    char const * const errId = "parallel:gpu:culapslppot:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    src = mxGetDoubles(prhs[0]);   /* source */
    targ = mxGetDoubles(prhs[1]);  /* target */
    x = mxGetDoubles(prhs[2]);     /* density */
    M = (int)(mxGetN(prhs[1]));
    N = (int)(mxGetN(prhs[0]));

    /* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 128;
    int blocksPerGrid;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    if ((nrhs != 3) || (nlhs != 1)) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    /* potential, no explicit lap slp mat */
    blocksPerGrid = (M + threadsPerBlock - 1) / threadsPerBlock;
    hipMalloc((void**)&d_src, 3*N*sizeof(double));
    hipMemcpy(d_src, src, 3*N*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_targ, 3*M*sizeof(double));
    hipMemcpy(d_targ, targ, 3*M*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_x, N*sizeof(double));
    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_y, M * sizeof(double));
    culapslppot<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_targ, d_x, d_y, N, M);

    /* Copy result back to host */
    plhs[0] = mxCreateDoubleMatrix((mwSize)M, (mwSize)1, mxREAL);
    y = mxGetDoubles(plhs[0]); /* 1st output */
    hipMemcpy(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost);

    /* Free GPU memory */
    hipFree(d_src);
    hipFree(d_targ);
    hipFree(d_x);
    hipFree(d_y);
}