#include "hip/hip_runtime.h"
/*
 * cuda lap slp pot & matlab mex interface
 * 11/15/23 Hai, do not form matrix explicitly
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
// #include "hipblas.h"
/* these are for matlab */
#include "mex.h"
#include "gpu/mxGPUArray.h"

#define THRESH 1e-15

/*
 * Device code
 */
void __global__ culapslppot(double const * const src,
                            double const * const targ,
                            double const * const x,
                            double * const y,
                            int const N,
                            int const M)
{
  /* Calculate the global linear index, assuming a 1-d grid. */
  int const i = blockDim.x * blockIdx.x + threadIdx.x;
  double dx, dy, dz, dd, threshsq;
  threshsq = THRESH*THRESH;
  if (i<M) {
    y[i] = 0.0;
    for (int j = 0; j < N; ++j) {
      dx = src[3*j]   - targ[3*i];
      dy = src[3*j+1] - targ[3*i+1];
      dz = src[3*j+2] - targ[3*i+2];
      dd = dx*dx + dy*dy + dz*dz;
      if (dd>threshsq){
        y[i] += x[j]*rsqrt(dd);
        // y[i] += x[j]/sqrt(dd);
      }
    }
  }
}

/*
 * Host code (just slp potential)
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
  /* Declare all variables.*/
  mxDouble *src, *targ, *x;  /* input: source, target, density */
  mxDouble *y;               /* output: potential */
  double *d_src, *d_targ;
  double *d_x, *d_y;
  float *curuntime;
  //double *d_A;
  int N, M;                  /* dimension */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  char const * const errId = "parallel:gpu:culapslppot:InvalidInput";
  char const * const errMsg = "Invalid input to MEX file.";
  
  src = mxGetDoubles(prhs[0]);   /* source */
  targ = mxGetDoubles(prhs[1]);  /* target */
  x = mxGetDoubles(prhs[2]);     /* density */
  M = (int)(mxGetN(prhs[1]));
  N = (int)(mxGetN(prhs[0]));
  
  /* Choose a reasonably sized number of threads for the block. */
  int const threadsPerBlock = 64;
  int blocksPerGrid;
  
  /* Initialize the MathWorks GPU API. */
  mxInitGPU();
  if ((nrhs != 3) || (nlhs != 2)) {
      mexErrMsgIdAndTxt(errId, errMsg);
  }
  
  /* potential, no explicit lap slp mat */
  blocksPerGrid = (M + threadsPerBlock - 1) / threadsPerBlock;
  hipMalloc((void**)&d_src, 3*N*sizeof(double));
  hipMemcpy(d_src, src, 3*N*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_targ, 3*M*sizeof(double));
  hipMemcpy(d_targ, targ, 3*M*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_x, N*sizeof(double));
  hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_y, M*sizeof(double));
  hipEventRecord(start);
  culapslppot<<<blocksPerGrid, threadsPerBlock>>>(d_src, d_targ, d_x, d_y, N, M);
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  /* Copy result back to host */
  plhs[0] = mxCreateDoubleMatrix((mwSize)M, (mwSize)1, mxREAL);
  y = mxGetDoubles(plhs[0]); /* 1st output */
  hipMemcpy(y, d_y, M*sizeof(double), hipMemcpyDeviceToHost);
  plhs[1] = mxCreateNumericMatrix(1,1,mxSINGLE_CLASS,mxREAL);
  curuntime = (float *) mxGetData(plhs[1]);
  curuntime[0] = milliseconds;
  
  /* Free GPU memory */
  hipFree(d_src);
  hipFree(d_targ);
  hipFree(d_x);
  hipFree(d_y);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}