#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h> // For the CUDA runtime routines (prefixed with "cuda_")
#include <math.h>
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;


#define TPB 16

using namespace std;

__global__ void MatBlockSum(double* A, double* blockSum, int N, int N0, int Nb){
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
__shared__ double temp[TPB][TPB];

if ((0 <= i) && (i < N0) && (0 <= j) && (j < N0)){
temp[threadIdx.y][threadIdx.x] = A[i+j*N];
} else{
temp[threadIdx.y][threadIdx.x] = 0;
}
__syncthreads();

int incr;
incr = TPB/2;
//__syncthreads();
if((threadIdx.x+incr < TPB)&&(threadIdx.y+incr < TPB)){
temp[threadIdx.y][threadIdx.x] = temp[threadIdx.y][threadIdx.x] + temp[threadIdx.y][threadIdx.x+incr] + temp[threadIdx.y+incr][threadIdx.x] + temp[threadIdx.y+incr][threadIdx.x+incr];
}
__syncthreads();

incr = incr/2;
if((threadIdx.x+incr < TPB/2)&&(threadIdx.y+incr < TPB/2)){
temp[threadIdx.y][threadIdx.x] = temp[threadIdx.y][threadIdx.x] + temp[threadIdx.y][threadIdx.x+incr] + temp[threadIdx.y+incr][threadIdx.x] + temp[threadIdx.y+incr][threadIdx.x+incr];
}
__syncthreads();

incr = incr/2;
if((threadIdx.x+incr < TPB/4)&&(threadIdx.y+incr < TPB/4)){
temp[threadIdx.y][threadIdx.x] = temp[threadIdx.y][threadIdx.x] + temp[threadIdx.y][threadIdx.x+incr] + temp[threadIdx.y+incr][threadIdx.x] + temp[threadIdx.y+incr][threadIdx.x+incr];
}
__syncthreads();

incr = incr/2;
if((threadIdx.x+incr < TPB/8)&&(threadIdx.y+incr < TPB/8)){
temp[threadIdx.y][threadIdx.x] = temp[threadIdx.y][threadIdx.x] + temp[threadIdx.y][threadIdx.x+incr] + temp[threadIdx.y+incr][threadIdx.x] + temp[threadIdx.y+incr][threadIdx.x+incr];
}
__syncthreads();

blockSum[blockIdx.x+blockIdx.y*Nb/TPB] = temp[0][0];
}

__global__ void MatMedian(double* A, double* B, int N, int N0){
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
__shared__ double temp[TPB+2][TPB+2];
if((threadIdx.x == TPB-1)&&(i < N0)){
temp[threadIdx.y+1][TPB+1] = A[i+1+j*N];
}
if((threadIdx.x == 0)&&(i >0)){
temp[threadIdx.y+1][0] = A[i-1+j*N];
}
if((threadIdx.y == 0)&&(j > 0)){
temp[0][threadIdx.x+1] = A[i+(j-1)*N];
}
if((threadIdx.y == TPB-1)&&(j < N0)){
temp[TPB+1][threadIdx.x+1] = A[i+(j+1)*N];
}
temp[threadIdx.y+1][threadIdx.x+1] = A[i+j*N];
__syncthreads();
if ((0 < i) && (i < N0-1) && (0 < j) && (j < N0-1)){
double *a = &temp[threadIdx.y+1][threadIdx.x+2];
double *b = &temp[threadIdx.y+1][threadIdx.x];
double *c = &temp[threadIdx.y][threadIdx.x+1];
double *d = &temp[threadIdx.y+2][threadIdx.x+1];
double *e = &temp[threadIdx.y+1][threadIdx.x+1];
double *tmp;
// makes a < b and b < d
if(*b < *a){
tmp = a; a = b; b = tmp;
}
if(*d < *c){
tmp = c; c = d; d = tmp;
}
// eleminate the lowest
if(*c < *a){
tmp = b; b = d; d = tmp;
c = a;
}
// gets e in
a = e;
// makes a < b and b < d
if(*b < *a){
tmp = a; a = b; b = tmp;
}
// eliminate another lowest
// remaing: a,b,d
if(*a < *c){
tmp = b; b = d; d = tmp;
a = c;
}
if(*d < *a)
B[i+j*N] = *d;
else
B[i+j*N] = *a;
} else if ((0 == i) || (i == N0-1) || (0 == j) || (j == N0-1)) {
B[i+j*N] = temp[threadIdx.y+1][threadIdx.x+1];
} else {
B[i+j*N] = 0;
}
}


int main(int argc, char *argv[]){
int N0;
for(int i=1; i<argc; i++){
N0 = atoi(argv[i]);
}
int N = (ceil((N0-1)/(4*TPB))+1)*TPB*4;
cout << "Number of elements each direction is " << N << endl;
double** a = new double*[N];
double** b = new double*[N];
double* blockSum = new double[((N-1)/TPB + 1)*((N-1)/TPB + 1)];

a[0] = new double[N*N]();
b[0] = new double[N*N];

for(int i=1; i<N; i++){
a[i] = a[i-1]+N;
b[i] = b[i-1]+N;
}

double* dev_a;
double* dev_b;
double* dev_blockSum;
double* dev_sum;

size_t pitch;
for (int i = 0; i < N0; ++i)
{
for (int j = 0; j < N0; ++j)
{
a[i][j] = pow(sin(i*i+j), 2) + cos(i-j);
}
}

hipMallocPitch(&dev_a,&pitch,sizeof(double)*N,N);
hipMallocPitch(&dev_b,&pitch,sizeof(double)*N,N);
hipMalloc((void **)&dev_blockSum, ((N-1)/TPB + 1)*((N-1)/TPB + 1)*sizeof(double));
hipMalloc((void **)&dev_sum, 1*sizeof(double));
hipMemcpy2D(dev_a,pitch,a[0],sizeof(double)*N,sizeof(double)*N,N,hipMemcpyHostToDevice);


dim3 gridDim((N-1)/TPB + 1, (N-1)/TPB + 1);
dim3 blockDim(TPB, TPB);
auto t1 = Clock::now();
for (int iter=0; iter<5; iter++){
MatMedian <<<gridDim, blockDim >>>(dev_a, dev_b, N, N0);
MatMedian <<<gridDim, blockDim >>>(dev_b, dev_a, N, N0);
}
auto t2 = Clock::now();
MatBlockSum <<<gridDim, blockDim >>>(dev_a, dev_blockSum, N, N0, N);
int flag = (N-1)/TPB;

while (flag > 0){
double* blockSum2 = new double[(flag/TPB+1)*(flag/TPB+1)*sizeof(double)];
double* dev_blockSum2;
hipMalloc((void **)&dev_blockSum2, (flag/TPB + 1)*(flag/TPB + 1)*sizeof(double));
gridDim.x = flag/TPB + 1;
gridDim.y = flag/TPB + 1;
MatBlockSum <<< gridDim, blockDim>>>(dev_blockSum, dev_blockSum2,flag + 1,flag + 1, (flag/TPB + 1)*TPB);
flag = flag/TPB;
dev_blockSum = dev_blockSum2;
}
auto t3 = Clock::now();
cout << "===========================================================" << endl;
std::cout <<"Iternation time taken: " << std::chrono::duration_cast<std::chrono::microseconds>(t2-t1).count() << "microseconds" << std::endl;
std::cout <<"Summation time taken: " << std::chrono::duration_cast<std::chrono::microseconds>(t3-t2).count() << "microseconds" << std::endl;
std::cout <<"Total time taken: " << std::chrono::duration_cast<std::chrono::microseconds>(t3-t1).count() << "microseconds" << std::endl;

hipMemcpy2D(b[0],sizeof(double)*N,dev_a,pitch,sizeof(double)*N,N,hipMemcpyDeviceToHost);
hipMemcpy(blockSum, dev_blockSum, (flag + 1)*(flag + 1)*sizeof(double),hipMemcpyDeviceToHost);

cout << endl;
cout << "Total sum is " << blockSum[0] << endl;

cout << "A(n/2,n/2) is " << b[N0/2][N0/2] << endl;
cout << "A(17,31) is " << b[17][31] << endl;

hipFree(dev_a);
hipFree(dev_b);
}
